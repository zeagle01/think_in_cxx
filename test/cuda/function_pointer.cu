#include "hip/hip_runtime.h"

#include "gmock/gmock.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>

using namespace testing;

namespace cuda_function_pointer
{

	template<typename T>
	using kernel_func_t = void (*)(T* a, T* b, int tid);

	template<typename T>
	__global__ void cuda_loop(kernel_func_t<T> kernel_func, T* a, T* b, int n)
	{
		int tid = threadIdx.x + blockDim.x * blockIdx.x;
		if (tid < n)
		{
			kernel_func(a, b, tid);
		}

	}

	template<typename T>
	__device__ void compute_kernel( T* a, T* b, int tid)
	{
		a[tid] = b[tid] * b[tid];
	}

	template<typename T>
	__device__ void compute_kernel_1( T* a, T* b, int tid)
	{
		a[tid] = b[tid] * b[tid] * b[tid];
	}

	template<typename T>
	__device__ kernel_func_t<T> d_pointer_compute_kernel = compute_kernel<T>;

	template<typename T>
	__device__ kernel_func_t<T> d_pointer_compute_kernel_1 = compute_kernel_1<T>;


	TEST(Cuda_Function_Pointer,use)
	{
		int num = 100;
		std::vector<float> h_a(num);
		std::vector<float> h_b(num);
		for (int i = 0; i < num; i++)
		{
			h_b[i] = i;
		}

		float* d_a;
		float* d_b;
		size_t size_in_byte = sizeof(float) * num;
		hipMalloc((void**)&d_a, size_in_byte);
		hipMalloc((void**)&d_b, size_in_byte);
		hipMemcpy(d_a, h_a.data(), size_in_byte, hipMemcpyHostToDevice);
		hipMemcpy(d_b, h_b.data(), size_in_byte, hipMemcpyHostToDevice);

		kernel_func_t<float> h_kernel_func;

		{
			hipMemcpyFromSymbol(&h_kernel_func, HIP_SYMBOL(d_pointer_compute_kernel<float>), sizeof(kernel_func_t<float>));

			int tpb = 128;
			int bpg = (num + tpb - 1) / tpb;
			cuda_loop << <bpg, tpb >> > (h_kernel_func, d_a, d_b, num);

			hipMemcpy(&h_a[0], d_a, size_in_byte, hipMemcpyDeviceToHost);
			for (int i = 0; i < num; i++)
			{
				EXPECT_THAT(h_a[i], Eq(i * i));
			}
		}

		{
			hipMemcpyFromSymbol(&h_kernel_func, HIP_SYMBOL(d_pointer_compute_kernel_1<float>), sizeof(kernel_func_t<float>));

			int tpb = 128;
			int bpg = (num + tpb - 1) / tpb;
			cuda_loop << <bpg, tpb >> > (h_kernel_func, d_a, d_b, num);

			hipMemcpy(&h_a[0], d_a, size_in_byte, hipMemcpyDeviceToHost);
			for (int i = 0; i < num; i++)
			{
				EXPECT_THAT(h_a[i], Eq(i * i * i));
			}
		}

	}


}