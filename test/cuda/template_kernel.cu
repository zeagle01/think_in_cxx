#include "hip/hip_runtime.h"

#include "gmock/gmock.h"
#include "hip/hip_runtime.h"
#include ""


namespace template_kernel
{

	template<typename T,int N>
	struct H_Vec 
	{
		T data[N];
	};

	template<typename T,int N>
	struct D_Vec 
	{
		T data[N];
	};




	template<typename T, int N, int Plat_Form>
	struct Vec;



	template<int ... N>
	static __device__ auto sum()
	{
		return sizeof...(N); //variadic template
	}



	template<template<typename, int > typename Vec_Type, typename T, int N> //template template
	__global__ void vec_add(Vec_Type<T, N>* v)
	{
		printf("hello %d\n", sum<1, 2, 3>());
	}



	TEST(Template_Kernel_Test,vector_add)
	{

		D_Vec<float,3> d;
		vec_add << <1, 1 >> > (&d);
	}

}