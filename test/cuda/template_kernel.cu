#include "hip/hip_runtime.h"

#include "gmock/gmock.h"
#include "hip/hip_runtime.h"
#include ""


namespace template_kernel
{

	template<typename T,int N>
	struct H_Vec 
	{
		T data[N];
	};

	template<typename T,int N>
	struct D_Vec 
	{
		T data[N];
	};




	template<typename T, int N, int Plat_Form>
	struct Vec;





	template<template<typename, int > typename Vec_Type, typename T, int N>
	__global__ void vec_add(Vec_Type<T, N>* v)
	{
		printf("hello\n");
	}



	TEST(Template_Kernel_Test,vector_add)
	{

		D_Vec<float,3> d;
		vec_add << <1, 1 >> > (&d);

	}

}