#include "hip/hip_runtime.h"

#include "gmock/gmock.h"
#include "hip/hip_runtime.h"
#include ""


namespace template_kernel
{

	template<typename T,int N>
	struct H_Vec 
	{
		T data[N];
	};

	template<typename T,int N>
	struct D_Vec 
	{
		T data[N];
	};




	template<typename T, int N, int Plat_Form>
	struct Vec;



	template<int ... N>
	static __device__ auto sum()
	{
		return sizeof...(N); //variadic template
	}



	template<template<typename, int > typename Vec_Type, typename T, int N> //template template
	__global__ void vec_add(Vec_Type<T, N>* v)
	{
		printf("hello %d\n", sum<1, 2, 3>());
	}



	TEST(Template_Kernel_Test,vector_add)
	{

		D_Vec<float,3> d;
		vec_add << <1, 1 >> > (&d);
	}


	template<typename T>
	struct B
	{
		T* p;
		int count;
	};

	struct A
	{
		B<int> p;
		float* p0;
		float* p1;
		float* p2;
		int n;
	};

	__global__ void vec_add(A a)
	{
		int tid = blockIdx.x * blockDim.x + threadIdx.x;
		if (tid < a.n)
		{
			a.p0[tid] = 2.f;
			printf("%d %f\n", a.n, a.p0[tid]);
		}

	}

	TEST(Template_Kernel_Test,pass_struct)
	{
		A a;

		a.n = 1;

		hipMalloc(&a.p0, a.n * sizeof(float));

		vec_add << <1, 1 >> > (a);
		hipError_t cudaerr = hipDeviceSynchronize();
		if (cudaerr != hipSuccess)
		{
			printf("kernel launch failed with error \"%s\".\n",
				hipGetErrorString(cudaerr));
		}

	}

}